#include<stdio.h>
#include<hip/hip_runtime.h>
#include<stdlib.h>

__global__ void hello(int* d_in){
  d_in[threadIdx.x] = (blockIdx.x + threadIdx.x) * (blockIdx.x * threadIdx.x);
  printf("%d : %d\n", threadIdx.x, d_in[threadIdx.x]);
}

int main(){
    int* h_array = (int*)malloc(1200*1024 * sizeof(int));
    int* d_array;
    hipMalloc((void**) &d_array, 1200*1024 * sizeof(int));
    /*for (int i = 0; i < 300; i++){
      h_array[i] = i;
    }*/
    hipMemcpy(d_array, h_array, 1200*1024*sizeof(int), hipMemcpyHostToDevice);

    hello<<<1200,1024>>>(d_array);
    hipMemcpy(h_array, d_array, sizeof(int) * 1200*1024, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    free(h_array);
    hipFree(d_array);
    return 0;
}
